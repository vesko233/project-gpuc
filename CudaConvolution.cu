#include "hip/hip_runtime.h"
#include "CudaConvolution.cuh"

// GRID DIMENSIONS = 3 x 16
// BLOCK DIMENSIONS = 32x32 = 1024
// blockIdx.y in the kernel below refers to the layer of the input tensor and filter, i.e. the z dimension
void convolution2D_kernel(float* d_image, float* d_filter, float* d_output)
{
    // Allocating shared memory in each block for the entire input image referring to this blockIdx
    // Each block of threads will be performing a 2D convolution of a layer of the input image with the corresponding layer of the filter
    // The grid will have dimensions 3 x 16, referring to the fact that we will perform convolutions with 16 3x3x3 filters.
    __shared__ float shared_image[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float shared_filter[FILTER_SIZE][FILTER_SIZE];

    // Fetching from global memory layer of image, depending on the blockIdx.y
    shared_image[threadIdx.y][threadIdx.x] = d_image[threadIdx.x + threadIdx.y*BLOCK_SIZE + blockIdx.y*BLOCK_SIZE*BLOCK_SIZE];

    // Fetching from global memory layer of filter, depending on the blockIdx.y
    if (threadIdx.x < FILTER_SIZE && threadIdx.y < FILTER_SIZE){
        shared_filter[threadIdx.y][threadIdx.x] = d_filter[threadIdx.x + threadIdx.y*FILTER_SIZE + blockIdx.y*FILTER_SIZE*FILTER_SIZE + blockIdx.x*FILTER_SIZE*FILTER_SIZE*DEPTH];
    }

    __syncthreads();

    float Res = 0;

    if (threadIdx.y < output_size && threadIdx.x < output_size){
        // Each thread performing convolution with filter
        for (int i = 0; i < FILTER_SIZE; i++){
            for (int j = 0; j < FILTER_SIZE; j++){
                Res += shared_image[threadIdx.y + i][threadIdx.x + j]*shared_filter[threadIdx.y][threadIdx.x];
            }
        }
        // Adding result to output. Here, we are using atomic add to avoid errors
        atomicAdd(&d_output[threadIdx.y + output_size*threadIdx.x + blockIdx.x*output_size*output_size],Res)
    }
}


// image = Flattened 3D tensor (32x32x3)
// filter = Flattened 4D tensor (3x3x3x16)
// output = Flattened 3D tensor (30x30x16)
void convolution_2D(float* image, float* filter, float* output)
{
    // 32 x 32 block
    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE,1);

    // 3 x 16 grid
    dim3 dimGrid(DEPTH,FILTERS_NUM,1);

    // Declaring device copies of data
    float* d_image;
    float* d_filter;
    float* d_output;

    // Initializing memory sizes for image, filter and output
    int image_memory_size = BLOCK_SIZE*BLOCK_SIZE*DEPTH*sizeof(float); // 32 x 32 x 3 x (float)
    int filter_memory_size = FILTER_SIZE*FILTER_SIZE*DEPTH*FILTERS_NUM*sizeof(float); // 3 x 3 x 3 x 16 x (float)
    int output_memory_size = (BLOCK_SIZE - FILTER_SIZE + 1)*(BLOCK_SIZE - FILTER_SIZE + 1)*FILTERS_NUM*sizeof(float); // 30 x 30 x 16 x (float)

    // Allocating memory on device
    hipMalloc((void**) &d_image, image_memory_size);
    hipMalloc((void**) &d_filter, filter_memory_size);
    hipMalloc((void**) &d_output, output_memory_size);

    // Copying memory on device from host
    hipMemcpy(d_image, image, image_memory_size, hipMemcpyHostToDevice);
    hipMemcpy(d_filter, filter, filter_memory_size, hipMemcpyHostToDevice);
    hipMemcpy(d_output, output, output_memory_size, hipMemcpyHostToDevice);
    
    // Executng the kernel
    convolution2D_kernel<<<dimGrid,dimBlock>>>(d_image,d_filter,d_output);

    // Copy result back to host memory from device memory
    hipMemcpy(output, d_output, output_memory_size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_image);
    hipFree(d_filter);
    hipFree(d_output);

    return 0;
}
