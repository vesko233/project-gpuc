#include "hip/hip_runtime.h"
#include "CudaGEMV.cuh"

__global__
void kernelST(float *vec, float *mat, float *b, float *out, const unsigned int N, const unsigned int M)
{
    int tid=threadIdx.x+blockIdx.x*blockDim.x;
    float sum=0.0f;
    if(tid<M){
        for(int i=0; i<N; i++)
            sum += vec[i]*mat[(tid*N)+i];
        out[tid]=sum + b[tid];
    }
    __syncthreads();
}

void matvec_kernel_cuda(float* input, float* matrix, float* biases, float* output,  unsigned int N, unsigned int M)
{
    float *dev_input, *dev_matrix, *dev_biases, *dev_output;

    hipMalloc((void**)&dev_input, sizeof(float)*N);
    hipMalloc((void**)&dev_matrix, sizeof(float)*N*M);
    hipMalloc((void**)&dev_biases, sizeof(float)*M);
    hipMalloc((void**)&dev_output, sizeof(float)*M);

    hipMemcpy(dev_input, input, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(dev_matrix, matrix, sizeof(float)*N*M, hipMemcpyHostToDevice);
    hipMemcpy(dev_biases, biases, sizeof(float)*M, hipMemcpyHostToDevice);

    kernelST<<<1,M>>>(dev_input, dev_matrix, dev_biases, dev_output, N, M);

    hipMemcpy(output, dev_output, sizeof(float)*M, hipMemcpyDeviceToHost);

    hipFree(dev_input);
    hipFree(dev_matrix);
    hipFree(dev_biases);
    hipFree(dev_output);
}




